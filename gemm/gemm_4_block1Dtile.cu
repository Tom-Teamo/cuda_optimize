#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <mma.h>

/*
    各个compute capability下的技术细节：
        https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications-technical-specifications-per-compute-capability

    SM内最大的常驻blocks：16
    SM内最大的常驻warp：48
    SM内最大的常驻thread：1536

    32-bit的寄存器数量：64k（k=1024）
    block内最多的32-bit寄存器数量：64k
    thread内最多的32-bit寄存器数量：255

    SM内最大的sMem：100KB
    block内最大的sMem：99KB
*/

/*
这要求M是BM的倍数，N是BN的倍数，K是BK的倍数

什么是 1维 tile？
    之前，需要计算C矩阵中的Bc:[BLOCK, BLOCK]大小的数据（因为矩阵大小未知 sharedMem有限 因此也需要分块）
        block中是需要 BLOCK* BLOCK 数量的线程的，每个线程计算Bc中的一个元素
    
    现在，需要计算C矩阵中的Bc:[BLOCK, BLOCK]大小的数据
        我们让每个线程读取[BLOCK, 1]大小的数据
        每个线程同样计算 [BLOCK, 1]大小的Bc的数据（下列代码每个线程计算Bc中的一列）
*/


#define OFFSET(row, col, stride) ((row) * (stride) + (col))
#define CEIL_DIV(M, N) (((M) + (N - 1)) / (N))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

template <const int BM, const int BN, const int BK, const int TM>
__global__ void tile_1d_kernel(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    
    __shared__ float As[BM][BK];
    __shared__ float Bs[BK][BN];
    float val[TM] = {0.};
    int num_shared_block = CEIL_DIV(K, BK); // or CEIL_DIV(K, BN);
    // 而&运算符被用来获取该区域的地址
    A = &A[OFFSET(blockIdx.y * BM, 0, K)];
    B = &B[OFFSET(0, blockIdx.x * BN, N)];
    C = &C[OFFSET(blockIdx.y * BM, blockIdx.x * BN, N)];

    for (int i = 0; i < num_shared_block; ++i) {
        // Copy data from global memory to shared memory
        for (int m = 0; m < TM; ++m) {
            int A_row = threadIdx.y * TM + m;
            int A_col = threadIdx.x;
            if ((blockIdx.y * BM + A_row) < M && (i * BK + A_col) < K) {
                As[A_row][A_col] = A[OFFSET(A_row, A_col, K)];
            } else {
                As[A_row][A_col] = 0.;
            }
        }
        int B_row = threadIdx.y;
        int B_col = threadIdx.x;
        if ((i * BK + B_row) < K && (blockIdx.x * BN + B_col) < N) {
            Bs[B_row][B_col] = B[OFFSET(B_row, B_col, N)];
        } else {
            Bs[B_row][B_col] = 0.;
        }
        __syncthreads();
        A += BK;
        B += BK * N;
        for (int k = 0; k < BK; ++k) {
            for (int m = 0; m < TM; ++m) { 
                int A_row = threadIdx.y * TM + m;
                int B_col = threadIdx.x;
                val[m] += As[A_row][k] * Bs[k][B_col];
            }
        }
        __syncthreads();
    }

    for (int m = 0; m < TM; ++m) {
        int C_row = threadIdx.y * TM + m;
        int C_col = threadIdx.x;
        if ((blockIdx.y * BM + C_row) < M && (blockIdx.x * BN + C_col) < N) {
            C[OFFSET(C_row, C_col, N)] = alpha * val[m] + beta * C[OFFSET(C_row, C_col, N)];
        }
    }
}


void MY_MMult(hipblasHandle_t handle, int M, int N, int K, float *A, int lda,
              float *B, int ldb, float *C, int ldc) {
    // 现在是什么情况呢
    // m n k 是要 整除 bm bn bk
    // BM 也必须整除 TM，BM TM是算法内部设计死的 所以只要代码里面满足就可以了 不像m n k 是用户输入的

    const int size = 16;
    const int tile_size = 8;
    const int BM = size * tile_size;
    const int BN = size;
    const int BK = size;
    const int TM = tile_size;

    const int alpha = 1.0;
    const int beta = 0.0;

    dim3 block(size, size);
    dim3 grid(CEIL_DIV(N, BN), CEIL_DIV(M, BM)); // note: change M and N here
    tile_1d_kernel<BM, BN, BK, TM><<<grid, block>>>(M, N, K, alpha, A, B, beta, C);
}